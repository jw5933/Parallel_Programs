#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RANGE 11.79

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU(float *ad, float *bd, float *cd, int n, int work, int total_threads);
/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){
	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	/*time adjustment:
	For the CUDA part in lab 3, the timing includes everything (cudamalloc, cudamemcpy and cudafree beside the kernel launch). 
	Those are very expensive operations and may not make the speed of GPU obvious. In real situations, we need to take them into account. 
	But, for this lab, you can adjust the given file to measure the time of the kernel launch only and not the cudamalloc, cudafree, and cudamemcpy.
	*/
	//1. allocate ad, bd, and cd in the device
	size_t SIZE = n*sizeof(float);
	hipError_t er;
	er = hipMalloc((void **)&ad, SIZE);
	if ( er != hipSuccess){
		printf("error w/ malloc %s\n", hipGetErrorString(er));
	};
	hipMalloc((void **)&bd, SIZE);
	hipMalloc((void **)&cd, SIZE);

	start = clock();
	
	//2. send a, b, and c to the device
	er = hipMemcpy(ad, a, SIZE, hipMemcpyHostToDevice);
	if ( er != hipSuccess){
		printf("error with memcpy %s\n", hipGetErrorString(er));
	};

	hipMemcpy(bd, b, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(cd, c, SIZE, hipMemcpyHostToDevice);

	//3. write the kernel, call it: vecGPU
	//4. Call the kernel (the kernel itself will be written at the comment at the end of this file), you need to write the number of threads, blocks, etc and their geometry.
	float num_threads = 500;
	int num_blocks = 4;
	int work = ceil(n/(num_threads*num_blocks));
	vecGPU<<<num_blocks, num_threads>>>(ad, bd, cd, n, work, num_threads*num_blocks);
	//cudaDeviceSynchronize();

	//5. Bring the cd array back from the device and store it in c array (declared earlier in main)
	er = hipMemcpy(c, cd, SIZE, hipMemcpyDeviceToHost);
	if ( er != hipSuccess){
		printf("error with memcpy d to h%s\n", hipGetErrorString(er));
	};

	end = clock();
	
	//6. free ad, bd, and cd
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);

	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( fabsf(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU(float *ad, float *bd, float *cd, int n, int work, int total_threads) {
	for (int w = 0; w < work; w++){
		int i = ((blockIdx.x * blockDim.x) + (threadIdx.x)) + w*total_threads;
		//printf("%d\n", i);
		if (i < n){
			cd[i]+= ad[i]*bd[i];
		}
	}
}